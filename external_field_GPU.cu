
#include <hip/hip_runtime.h>
/********************************************************************************
*
*   Copyright (C) 2015 Culham Centre for Fusion Energy,
*   United Kingdom Atomic Energy Authority, Oxfordshire OX14 3DB, UK
*
*   Licensed under the Apache License, Version 2.0 (the "License");
*   you may not use this file except in compliance with the License.
*   You may obtain a copy of the License at
*
*       http://www.apache.org/licenses/LICENSE-2.0
*
*   Unless required by applicable law or agreed to in writing, software
*   distributed under the License is distributed on an "AS IS" BASIS,
*   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
*   See the License for the specific language governing permissions and
*   limitations under the License.
*
********************************************************************************
*
*   Program: SPILADY - A Spin-Lattice Dynamics Simulation Program
*   Version: 1.0
*   Date:    Aug 2015
*   Author:  Pui-Wai (Leo) MA
*   Contact: info@spilady.ccfe.ac.uk
*   Address: Culham Centre for Fusion Energy, OX14 3DB, United Kingdom
*
********************************************************************************/

#if defined GPU

#include "spilady.h"
#include "prototype_GPU.h"

#if defined extfield

#include "prototype_GPU.h"

/************************************************************************
* GPU prototypes
************************************************************************/

__global__ void LP1extfield(struct varGPU *var_ptr_d, struct atom_struct *first_atom_ptr_d);


/************************************************************************
* CPU codes
************************************************************************/

void external_field_GPU(int current_step){

    static bool infile_extfield = 0;

    if (current_step ==  -1){
        ifstream infile("extfield.in");

        if (infile) {
            cout << "Reading external field file!!!" << '\n';
            infile_extfield = 1;

            int temp;
            infile >> natom;
            for (int i = 0; i < natom; ++i){
                struct atom_struct* atom_ptr;
                atom_ptr = first_atom_ptr + i;
                infile >> temp >> atom_ptr->Hext.x >> atom_ptr->Hext.y >> atom_ptr->Hext.z;
                atom_ptr->Hext = vec_times(-el_g*muB, atom_ptr->Hext);  // converted from Tesla into eV

            }
        }
    }

    if (infile_extfield == 0){

        if (current_step == 0) cout <<  "User defined external field apply." << '\n';

        Hext.x = 0e0; //in Tesla
        Hext.y = 0e0; //in Tesla
        Hext.z = 0e0; //in Tesla

        //if (total_time < 1e-10) Hext.z = 0e0;
        //if (total_time >= 1e-10 && total_time < 4e-10) Hext.z = -20e0;
        //if (total_time >= 4e-10 && total_time < 7e-10) Hext.z = 0e0;
        //if (total_time >= 7e-10) Hext.z = -20e0;

        Hext = vec_times(-el_g*muB, Hext);  // converted into eV

        cudaMemcpy(&(var_ptr_d->Hext), &Hext, sizeof(vector), cudaMemcpyHostToDevice);
        LP1extfield<<<no_of_blocks, no_of_threads>>>(var_ptr_d, first_atom_ptr_d);
    }
}

void external_field(int current_step){
    external_field_GPU(current_step);
}


/**************************************************************************************
* GPU codes
**************************************************************************************/

__global__ void LP1extfield(struct varGPU *var_ptr_d, struct atom_struct *first_atom_ptr_d){

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < var_ptr_d->natom)  (first_atom_ptr_d + i)->Hext = var_ptr_d->Hext;

}

#endif
#endif
